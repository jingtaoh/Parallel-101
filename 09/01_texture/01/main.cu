#include "hip/hip_runtime.h"
#include <cstdio>
#include <vector>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include "hip/hip_vector_types.h"
#include "CudaArray.cuh"
#include "ticktock.h"
#include "writevdb.h"

__global__ void advect_kernel(CudaTexture<float4>::Accessor texVel, CudaSurface<float4>::Accessor sufLoc, unsigned int n) {
    unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;
    unsigned int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= n || y >= n || z >= n) return;

    float4 vel = texVel.sample(x, y, z);
    float4 loc = make_float4(x + 0.5f, y + 0.5f, z + 0.5f, 42.f) - vel;
    sufLoc.write(loc, x, y, z);
}

__global__ void resample_kernel(CudaSurface<float4>::Accessor sufLoc, CudaTexture<float4>::Accessor texClr, CudaSurface<float4>::Accessor sufClrNext, unsigned int n) {
    unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;
    unsigned int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= n || y >= n || z >= n) return;

    float4 loc = sufLoc.read(x, y, z);
    float4 clr = texClr.sample(loc.x, loc.y, loc.z);
    sufClrNext.write(clr, x, y, z);
}

template <class T>
struct CudaAST {
    CudaArray<T> arr;
    CudaSurface<T> suf;
    CudaTexture<T> tex;

    CudaAST(ctor_t, typename CudaArray<T>::BuildArgs const &_arrArgs, typename CudaTexture<T>::BuildArgs const &_texArgs = {})
        : arr(ctor, _arrArgs)
        , suf(ctor, arr)
        , tex(ctor, arr, _texArgs)
    {
    }
};

struct SmokeSim {
    nocopy_t nocopy;

    unsigned int n;
    CudaAST<float4> loc;
    CudaAST<float4> vel;
    CudaAST<float4> velNext;
    CudaAST<float4> clr;
    CudaAST<float4> clrNext;

    SmokeSim(unsigned int _n)
    : n(_n)
    , loc(ctor, {{n, n, n}})
    , vel(ctor, {{n, n, n}})
    , velNext(ctor, {{n, n, n}})
    , clr(ctor, {{n, n, n}})
    , clrNext(ctor, {{n, n, n}})
    {}

    void advection() {
        advect_kernel<<<dim3((n + 7) / 8, (n + 7) / 8, (n + 7) / 8), dim3(8, 8, 8)>>>(vel.tex.access(), loc.suf.access(), n);
        resample_kernel<<<dim3((n + 7) / 8, (n + 7) / 8, (n + 7) / 8), dim3(8, 8, 8)>>>(loc.suf.access(), clr.tex.access(), clrNext.suf.access(), n);
        resample_kernel<<<dim3((n + 7) / 8, (n + 7) / 8, (n + 7) / 8), dim3(8, 8, 8)>>>(loc.suf.access(), vel.tex.access(), velNext.suf.access(), n);

        std::swap(vel, velNext);
        std::swap(clr, clrNext);
    }
};

int main() {
    unsigned int n = 64;

    SmokeSim sim(n);

    {
        std::vector<float4> cpu(n * n * n);
        for (unsigned int z = 0; z < n; z++) {
            for (unsigned int y = 0; y < n; y++) {
                for (unsigned int x = 0; x < n; x++) {
                    float den = std::hypot((int)x - (int)n / 2, (int)y - (int)n / 2, (int)z - (int)n / 2) < n / 2 ? 1.f : 0.f;
                    cpu[x + n * (y + n * z)] = make_float4(den, 0.f, 0.f, 0.f);
                }
            }
        }
        sim.clr.arr.copyIn(cpu.data());
    }

    {
        std::vector<float4> cpu(n * n * n);
        for (unsigned int z = 0; z < n; z++) {
            for (unsigned int y = 0; y < n; y++) {
                for (unsigned int x = 0; x < n; x++) {
                    cpu[x + n * (y + n * z)] = make_float4(0.1f, 0.f, 0.f, 0.f);
                }
            }
        }
        sim.vel.arr.copyIn(cpu.data());
    }

    std::vector<float4> cpu(n * n * n);
    for (int frame = 1; frame <= 100; frame++) {
        sim.clr.arr.copyOut(cpu.data());
        writevdb<float, 1>("/tmp/a" + std::to_string(1000 + frame).substr(1) + ".vdb", cpu.data(), n, n, n, sizeof(float4));

        printf("frame=%d\n", frame);
        sim.advection();
    }

    return 0;
}
